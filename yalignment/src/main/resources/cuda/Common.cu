#include "hip/hip_runtime.h"
#include "Common.h"

__device__ void scoresTracebackStep(
        short primaryItem,
        short substitutionItem,
        short verticalItem,
        short horizontalItem,
        char symbol1,
        char symbol2,
        int* i,
        int* j,
        int* matchesCount) {

    if (primaryItem == substitutionItem) {
        if (symbol1 == symbol2) *matchesCount += 1;
        *i -= 1;
        *j -= 1;
    }
    else if (primaryItem == verticalItem)
        *i -= 1;
    else if (primaryItem == horizontalItem)
        *j -= 1;
}

__device__ void initialNil(
        short* primary,
        short* substitution,
        short* vertical,
        short* horizontal,
        int idx) {

    horizontal[idx] = vertical[idx] = substitution[idx] = primary[idx] = 0;
}

__device__ void setMatricesValues(
        short* primary,
        short* substitution,
        short* vertical,
        short* horizontal,
        int currentIdx,
        int previousHorIdx,
        int previousVerIdx,
        int previousDiagIdx,
        int gapExtendPenalty,
        int gapOpenPenalty,
        char horizontalItem,
        char verticalItem,
        short* substitutionScores) {

    short horItem = horizontal[currentIdx] = max(
        horizontal[previousHorIdx] + gapExtendPenalty,
        primary[previousHorIdx] + gapOpenPenalty
    );
    short verItem = vertical[currentIdx] = max(
        vertical[previousVerIdx] + gapExtendPenalty,
        primary[previousVerIdx] + gapOpenPenalty
    );
    short subsItem = substitution[currentIdx] = primary[previousDiagIdx] +
        substitutionScores[horizontalItem * charSize + verticalItem];

    primary[currentIdx] = max(subsItem, max(verItem, horItem));
}
