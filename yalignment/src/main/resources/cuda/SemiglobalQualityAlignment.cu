#include "hip/hip_runtime.h"
#include "Common.h"

__device__ char* setChar(char *dest, const char ch, int offset, int position) {
    dest[offset + position] = ch;
    return dest;
}

__device__ void calculateSemiglobalQualityAlignmentValues(
        int i,
        int j,
        int k,
        char horizontalItem,
        char verticalItem,
        short* primary,
        short* substitution,
        short* horizontal,
        short* vertical,
        short* substitutionScores,
        int size,
        int gapOpenPenalty,
        int gapExtendPenalty) {

    int pairOffset = k * size * size;
    int currentIdx = pairOffset + i * size + j;

    if (i == 0 || j == 0) {
        initialNil(primary, substitution, vertical, horizontal, currentIdx);

        return;
    }

    int previousHorIdx = currentIdx - 1;
    int previousVerIdx = pairOffset + (i - 1) * size + j;
    int previousDiagIdx = previousVerIdx - 1;

    setMatricesValues(
        primary,
        substitution,
        vertical,
        horizontal,
        currentIdx,
        previousHorIdx,
        previousVerIdx,
        previousDiagIdx,
        gapExtendPenalty,
        gapOpenPenalty,
        horizontalItem,
        verticalItem,
        substitutionScores);
}

__device__ char* consensusItem(char aQuality, char bQuality, char a, char b) {
    char result[2] = {
        (aQuality >= bQuality) ? a : b,
        max(aQuality, bQuality)
    };

    return result;
}

__device__ void consensuses(
        int k,
        char* a,
        char* b,
        int maxSize,
        short fRealSize,
        short rRealSize,
        short* primary,
        short* substitution,
        short* horizontal,
        short* vertical,
        char* aQualities,
        char* bQualities,
        char gapQuality,
        char* consensus,
        char* consensusQualities,
        short* scores) {

    int pairOffset = k * maxSize * maxSize;
    int readOffset = k * (maxSize -1);
    int consensusOffset = readOffset * 2;
    short n;

    short verMax = 0, verMaxIndex = 0, horMax = 0, horMaxIndex = 0;
    for (n = 0; n < rRealSize + 1; n++) {
        short currentVer = primary[pairOffset + n * maxSize + fRealSize];
        if (currentVer > verMax) {
            verMax = currentVer;
            verMaxIndex = n;
        }
    }
    for (n = 0; n < fRealSize + 1; n++) {
        short currentHor = primary[pairOffset + rRealSize * maxSize + n];
        if (currentHor > horMax) {
            horMax = currentHor;
            horMaxIndex = n;
        }
    }

    int i = rRealSize, j = fRealSize;
    short mCon = 0, mQual = 0;

    //traceback end
    if (horMax >= verMax) { //deletions
        for (n = 0; n < fRealSize - horMaxIndex; n++) {
            char aQ = aQualities[readOffset + j - n - 1];
            if (aQ >= gapQuality) {
                setChar(consensus, a[readOffset + j - n - 1], consensusOffset, mCon++);
                setChar(consensusQualities, aQ, consensusOffset, mQual++);
            }
        }
        j = horMaxIndex;
    }
    else { //insertions
        for (n = 0; n < rRealSize - verMaxIndex; n++) {
            char bQ = bQualities[readOffset + i - n - 1];
            if (bQ > gapQuality) {
                setChar(consensus, b[readOffset + i - n - 1], consensusOffset, mCon++);
                setChar(consensusQualities, bQ, consensusOffset, mQual++);
            }
        }
        i = verMaxIndex;
    }

    //traceback middle
    while (i > 0 && j > 0) {
        short primaryItem = primary[pairOffset + i * maxSize + j];

        if (primaryItem == substitution[pairOffset + i * maxSize + j]) { //match or mismatch
            char* _consensusItem = consensusItem(
                                    aQualities[readOffset + j - 1],
                                    bQualities[readOffset + i - 1],
                                    a[readOffset + j - 1],
                                    b[readOffset + i - 1]);
            setChar(consensus, _consensusItem[0], consensusOffset, mCon++);
            setChar(consensusQualities, _consensusItem[1], consensusOffset, mQual++);

            i -= 1;
            j -= 1;
        }
        else if (primaryItem == vertical[pairOffset + i * maxSize + j]) { //insertion
            char bQ = bQualities[readOffset + i - 1];
            if (bQ > gapQuality) {
                setChar(consensus, b[readOffset + i - 1], consensusOffset, mCon++);
                setChar(consensusQualities, bQ, consensusOffset, mQual++);
            }

            i -= 1;
        }
        else if (primaryItem == horizontal[pairOffset + i * maxSize + j]) { //deletion
            char aQ = aQualities[readOffset + j - 1];
            if (aQ >= gapQuality) {
                setChar(consensus, a[readOffset + j - 1], consensusOffset, mCon++);
                setChar(consensusQualities, aQ, consensusOffset, mQual++);
            }

            j -= 1;
        }
    }

    //traceback start
    if (i == 0) { //deletions
        for (n = 0; n < j; n++) {
            char aQ = aQualities[readOffset + j - n - 1];
            if (aQ >= gapQuality) {
                setChar(consensus, a[readOffset + j - n - 1], consensusOffset, mCon++);
                setChar(consensusQualities, aQ, consensusOffset, mQual++);
            }
        }
    }
    else if (j == 0) { //insertion
        for (n = 0; n < i; n++) {
            char bQ = bQualities[readOffset + i - n - 1];
            if (bQ > gapQuality) {
                setChar(consensus, b[readOffset + i - n - 1], consensusOffset, mCon++);
                setChar(consensusQualities, bQ, consensusOffset, mQual++);
            }
        }
    }

    for (n = mCon; n < (maxSize - 1) * 2; n++) {
        setChar(consensus, '*', consensusOffset, mCon++);
        setChar(consensusQualities, 0, consensusOffset, mQual++);
    }

    scores[k] = max(horMax, verMax);
}

extern "C"
__global__ void semiGlobalQualityAlignment(
        int pairsCount,
        char* a,
        char* b,
        short* primary,
        short* substitution,
        short* horizontal,
        short* vertical,
        short* substitutionScores,
        int maxSize,
        short* realSizes,
        int gapOpenPenalty,
        int gapExtendPenalty,
        char* aQualities,
        char* bQualities,
        char gapQuality,
        char* consensus,
        char* consensusQualities,
        short* scores) {

    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k >= pairsCount)
        return;

    int readMaxSize = maxSize - 1;
    int fRealSize = realSizes[k * 2];
    int rRealSize = realSizes[k * 2 + 1];
    for (int i = 0; i < rRealSize + 1; i++) {
        for (int j = 0; j < fRealSize + 1; j++) {
            calculateSemiglobalQualityAlignmentValues(
                i,
                j,
                k,
                a[k * readMaxSize + j - 1],
                b[k * readMaxSize + i - 1],
                primary,
                substitution,
                horizontal,
                vertical,
                substitutionScores,
                maxSize,
                gapOpenPenalty,
                gapExtendPenalty);
        }
    }

    consensuses(
        k,
        a,
        b,
        maxSize,
        fRealSize,
        rRealSize,
        primary,
        substitution,
        horizontal,
        vertical,
        aQualities,
        bQualities,
        gapQuality,
        consensus,
        consensusQualities,
        scores);
}
