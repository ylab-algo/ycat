#include "hip/hip_runtime.h"
#include "GlobalAffineAlignment.h"
#include "LocalAffineAlignment.h"
#include "SemiglobalAffineAlignment.h"

__device__ void alignmentInternal(
        void (*calculateValuesFunc)(int,int,int,int,int,int,char,char,short*,short*,short*,short*,short*,int,int),
        void (*scoresFunc)(int,int,int,int,short*,short*,short*,short*,int,int,int,int,char*,char*,int,float*),
        int seqsCount,
        int refsCount,
        char* seqs,
        char* refs,
        short* seqSizes,
        int seqsMaxSize,
        short* refSizes,
        int refsMaxSize,
        short* primary,
        short* substitution,
        short* horizontal,
        short* vertical,
        short* substitutionScores,
        short gapOpenPenalty,
        short gapExtendPenalty,
        float* similarityScores) {

    int seqIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int refIdx = blockIdx.y * blockDim.y + threadIdx.y;

    if (seqIdx >= seqsCount)
        return;
    else {
        int seqMatrixSize = seqsMaxSize + 1;
        int refMatrixSize = refsMaxSize + 1;
        int matrixSize = seqMatrixSize * refMatrixSize;
        int seqOffset = seqIdx * refsCount * matrixSize;
        int refOffset = refIdx * matrixSize;
        int seqRefOffset = seqOffset + refOffset;

        for (int i = 0; i < seqSizes[seqIdx] + 1; i++) {
            for (int j = 0; j < refSizes[refIdx] + 1; j++) {
                int currentIdx = seqRefOffset + i * refMatrixSize + j;
                int previousHorIdx = currentIdx - 1;
                int previousVerIdx = seqRefOffset + (i - 1) * refMatrixSize + j;
                int previousDiagIdx = previousVerIdx - 1;

                calculateValuesFunc(
                    i,
                    j,
                    currentIdx,
                    previousHorIdx,
                    previousVerIdx,
                    previousDiagIdx,
                    refs[refIdx * refsMaxSize + j - 1],
                    seqs[seqIdx * seqsMaxSize + i - 1],
                    primary,
                    substitution,
                    horizontal,
                    vertical,
                    substitutionScores,
                    gapOpenPenalty,
                    gapExtendPenalty);
            }
        }

        scoresFunc(
            seqSizes[seqIdx],
            refSizes[refIdx],
            seqRefOffset,
            refMatrixSize,
            primary,
            substitution,
            horizontal,
            vertical,
            seqIdx,
            refIdx,
            seqsMaxSize,
            refsMaxSize,
            seqs,
            refs,
            refsCount,
            similarityScores);
    }
}

extern "C"
__global__ void alignment(
        char algorithmType,
        int seqsCount,
        int refsCount,
        char* seqs,
        char* refs,
        short* seqSizes,
        int seqsMaxSize,
        short* refSizes,
        int refsMaxSize,
        short* primary,
        short* substitution,
        short* horizontal,
        short* vertical,
        short* substitutionScores,
        short gapOpenPenalty,
        short gapExtendPenalty,
        float* similarityScores) {

    void (*calculateValuesFunc)(int,int,int,int,int,int,char,char,short*,short*,short*,short*,short*,int,int);
    void (*scoresFunc)(int,int,int,int,short*,short*,short*,short*,int,int,int,int,char*,char*,int,float*);
    if (algorithmType == 'l') {
        calculateValuesFunc = &calculateLocalValues;
        scoresFunc = &localSimilarityScores;
    }
    else if (algorithmType == 'g') {
        calculateValuesFunc = &calculateGlobalValues;
        scoresFunc = &globalSimilarityScores;
    }
    else {
        calculateValuesFunc = &calculateSemiglobalValues;
        scoresFunc = &semiglobalSimilarityScores;
    }

    alignmentInternal(
        calculateValuesFunc,
        scoresFunc,
        seqsCount,
        refsCount,
        seqs,
        refs,
        seqSizes,
        seqsMaxSize,
        refSizes,
        refsMaxSize,
        primary,
        substitution,
        horizontal,
        vertical,
        substitutionScores,
        gapOpenPenalty,
        gapExtendPenalty,
        similarityScores);
}
