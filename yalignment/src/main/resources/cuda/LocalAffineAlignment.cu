#include "hip/hip_runtime.h"
#include "Common.h"

__device__ void calculateLocalValues(
        int i,
        int j,
        int currentIdx,
        int previousHorIdx,
        int previousVerIdx,
        int previousDiagIdx,
        char horizontalItem,
        char verticalItem,
        short* primary,
        short* substitution,
        short* horizontal,
        short* vertical,
        short* substitutionScores,
        int gapOpenPenalty,
        int gapExtendPenalty) {

    if (i == 0 || j == 0) {
        initialNil(primary, substitution, vertical, horizontal, currentIdx);

        return;
    }

    setMatricesValues(
        primary,
        substitution,
        vertical,
        horizontal,
        currentIdx,
        previousHorIdx,
        previousVerIdx,
        previousDiagIdx,
        gapExtendPenalty,
        gapOpenPenalty,
        horizontalItem,
        verticalItem,
        substitutionScores);
    primary[currentIdx] = max(0, primary[currentIdx]);
}

__device__ void localSimilarityScores(
        int seqSize,
        int refSize,
        int seqRefOffset,
        int refMatrixSize,
        short* primary,
        short* substitution,
        short* horizontal,
        short* vertical,
        int seqIdx,
        int refIdx,
        int seqsMaxSize,
        int refsMaxSize,
        char* seqs,
        char* refs,
        int refsCount,
        float* similarityScores) {

    int currentIdx, i, j, matchesCount = 0;
    short max = negInfinity, maxI, maxJ;

    for (i = 0; i < seqSize + 1; i++) {
        for (j = 0; j < refSize + 1; j++) {
            currentIdx = seqRefOffset + i * refMatrixSize + j;
            if (primary[currentIdx] > max) {
                max = primary[currentIdx];
                maxI = i;
                maxJ = j;
            }
        }
    }

    i = maxI;
    j = maxJ;
    currentIdx = seqRefOffset + i * refMatrixSize + j;

    while (i != 0 && j != 0 && primary[currentIdx] != 0) {
        scoresTracebackStep(
            primary[currentIdx],
            substitution[currentIdx],
            vertical[currentIdx],
            horizontal[currentIdx],
            refs[refIdx * refsMaxSize + j - 1],
            seqs[seqIdx * seqsMaxSize + i - 1],
            &i,
            &j,
            &matchesCount);

        currentIdx = seqRefOffset + i * refMatrixSize + j;
    }

    similarityScores[seqIdx * refsCount + refIdx] = (float)matchesCount / seqSize;
}
